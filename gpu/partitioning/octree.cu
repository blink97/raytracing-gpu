#include "hip/hip_runtime.h"
#include "octree.h"

#include "sort.h"
#include "prefix_sum.h"

__device__ __forceinline__ float atomicMinFloat(float *addr, float value)
{
  float old = ((value >= 0)
    ? __int_as_float(atomicMin((int *)addr, __float_as_int(value)))
    : __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value))));

  return old;
}

__device__ __forceinline__ float atomicMaxFloat(float *addr, float value) {
  float old = ((value >= 0)
    ? __int_as_float(atomicMax((int *)addr, __float_as_int(value)))
    : __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value))));

    return old;
}

/*
 * Get the position of a point in one dimension
 * into a virtual octree. The given value must
 * lie in the [0-1] range
 */
__device__ uint8_t get_point_position(float value)
{
  return min((uint16_t)((value) * 256), 255);
}

/*
 * Get the level associated with the given position
 */
__device__ uint8_t get_level(octree_generation_position position)
{
  return position & 0xFF;
}

/*
 * Get the position in the octree node at the given level.
 */
__device__ uint8_t get_level_position(octree_generation_position position, uint8_t level)
{
  return (position >> (8/* Skip the level */ + 3 * (8 - level))) & 0x7;
}


__device__ uint8_t get_common_level(
    octree_generation_position first,
    octree_generation_position second)
{
  uint8_t min_level = min(get_level(first), get_level(second));
  uint8_t common_level = 0;
  for (; common_level < min_level; ++common_level)
  {
    if (get_level_position(first, common_level + 1) != get_level_position(second, common_level + 1))
    {// first and second levels are differents
      break;
    }
  }
  return common_level;
}

__global__ void find_scene_scale_basic(
  const struct AABB *const aabbs,
  size_t nb_objects,
  struct AABB *resulting_scale)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nb_objects) return; // Nothing to do here

  atomicMinFloat(&resulting_scale->min.x, aabbs[index].min.x);
  atomicMinFloat(&resulting_scale->min.y, aabbs[index].min.y);
  atomicMinFloat(&resulting_scale->min.z, aabbs[index].min.z);

  atomicMaxFloat(&resulting_scale->max.x, aabbs[index].max.x);
  atomicMaxFloat(&resulting_scale->max.y, aabbs[index].max.y);
  atomicMaxFloat(&resulting_scale->max.z, aabbs[index].max.z);
}

__global__ void find_scene_scale_shared(
  const struct AABB *const aabbs,
  size_t nb_objects,
  struct AABB *resulting_scale)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nb_objects) return; // Nothing to do here

  __shared__ struct AABB shared_scale;

  // Set the default value of the scale
  if (threadIdx.x == 0)
  {
    shared_scale = aabbs[index];
  }
  __syncthreads();

  atomicMinFloat(&shared_scale.min.x, aabbs[index].min.x);
  atomicMinFloat(&shared_scale.min.y, aabbs[index].min.y);
  atomicMinFloat(&shared_scale.min.z, aabbs[index].min.z);

  atomicMaxFloat(&shared_scale.max.x, aabbs[index].max.x);
  atomicMaxFloat(&shared_scale.max.y, aabbs[index].max.y);
  atomicMaxFloat(&shared_scale.max.z, aabbs[index].max.z);

  // Make sure that all the thread all computed the correct partial scale
  // before doing it at the global level.
  __syncthreads();

  if (threadIdx.x == 0)
  {
    atomicMinFloat(&resulting_scale->min.x, shared_scale.min.x);
    atomicMinFloat(&resulting_scale->min.y, shared_scale.min.y);
    atomicMinFloat(&resulting_scale->min.z, shared_scale.min.z);

    atomicMaxFloat(&resulting_scale->max.x, shared_scale.max.x);
    atomicMaxFloat(&resulting_scale->max.y, shared_scale.max.y);
    atomicMaxFloat(&resulting_scale->max.z, shared_scale.max.z);
  }
}

__device__ void scale_position(vector3 *position, const struct AABB *const scale)
{
  position->x = (position->x + scale->min.x) / (scale->max.x - scale->min.x);
  position->y = (position->y + scale->min.y) / (scale->max.y - scale->min.y);
  position->z = (position->z + scale->min.z) / (scale->max.z - scale->min.z);
}

// Get the level associated with the object
// (the octree node that can contains both min and max value)
__device__ void object_level(uint8_t min, uint8_t max, uint8_t *level)
{
  uint8_t current_level = 0;

  // Trying to find it's place from top to bottom:
  // If a level is accepted, it goes to the next one.
  while (current_level < 8 && (min & (1 << (7 - current_level))) == (max & (1 << (7 - current_level))))
  {
    ++current_level;
  }

  *level = current_level;
}

__global__ void position_object(
  const struct AABB *const aabbs,
  const struct AABB *const scale,
  octree_generation_position *positions,
  size_t nb_objects)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nb_objects) return; // Nothing to do here

  struct AABB current_aabb = aabbs[index];

  // Scale the AABB so that it is in the [0-1] cube
  scale_position(&current_aabb.min, scale);
  scale_position(&current_aabb.max, scale);

  // Start to find it's position.
  // To do so, lay a fixed grid, of the biggest depth and lay the number in it.
  // Doing so allows to have the correct value.
  // (1 if above center, 0 if under, whatever is the depth).

  uint8_t position_min_x = get_point_position(current_aabb.min.x);
  uint8_t position_min_y = get_point_position(current_aabb.min.y);
  uint8_t position_min_z = get_point_position(current_aabb.min.z);

  uint8_t position_max_x = get_point_position(current_aabb.max.x);
  uint8_t position_max_y = get_point_position(current_aabb.max.y);
  uint8_t position_max_z = get_point_position(current_aabb.max.z);

  uint8_t level_x, level_y, level_z;

  object_level(position_min_x, position_max_x, &level_x);
  object_level(position_min_y, position_max_y, &level_y);
  object_level(position_min_z, position_max_z, &level_z);

  // The final level is the top level in all axes.
  uint8_t final_level = min(min(level_x, level_y), level_z);

  // Compute the final position, position_min is used,
  // but position_min and position_max point to the same thing,
  // as the level is how many common bits they have in common.
  uint8_t resulting_position_x = position_max_x & (0xFF << (8 - final_level));
  uint8_t resulting_position_y = position_max_y & (0xFF << (8 - final_level));
  uint8_t resulting_position_z = position_max_z & (0xFF << (8 - final_level));

  // Create the resulting position.
  octree_generation_position position;
  for (int i = 0; i < 8; ++i)
  {
    position <<= 3;
    position |= ((resulting_position_x & (1 << (7 - i))) != 0 ? 1 : 0);
    position |= ((resulting_position_y & (1 << (7 - i))) != 0 ? 2 : 0);
    position |= ((resulting_position_z & (1 << (7 - i))) != 0 ? 4 : 0);
  }
  position = ((position << 8) | final_level);

  // And save it
  positions[index] = position;
}


__global__ void nodes_difference_array(
  const octree_generation_position *const sorted_positions,
  size_t *nodes_difference,
  size_t nb_objects)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nb_objects) return; // Nothing to do here

  octree_generation_position current = sorted_positions[index];
  uint8_t current_level = get_level(current);

  size_t diff;
  if (index == 0)
  {// The first node creates everything.
    diff = current_level + 1/* Include the root nodes */;
  }
  else
  {// The next nodes only created was is needed compared to the previous one.
    octree_generation_position previous = sorted_positions[index - 1];
    uint8_t previous_level = get_level(previous);

    // Don't include the root node, it already was included.
    // Remove the common level nodes in it.
    uint8_t common_level = get_common_level(previous, current);
    diff = current_level - common_level;
  }

  nodes_difference[index] = diff;
}


__device__ void get_aabb_box(
  uint8_t x, uint8_t y, uint8_t z, uint8_t level,
  const struct AABB *const scale, struct AABB *octree_aabb)
{
  float aabb_size = pow(0.5, level);
  octree_aabb->min.x = ((float)(x / 256.0)) * (scale->max.x - scale->min.x) - scale->min.x;
  octree_aabb->min.y = ((float)(y / 256.0)) * (scale->max.y - scale->min.y) - scale->min.y;
  octree_aabb->min.z = ((float)(z / 256.0)) * (scale->max.z - scale->min.z) - scale->min.z;

  octree_aabb->max.x = ((float)(x / 256.0) + aabb_size) * (scale->max.x - scale->min.x) - scale->min.x;
  octree_aabb->max.y = ((float)(y / 256.0) + aabb_size) * (scale->max.y - scale->min.y) - scale->min.y;
  octree_aabb->max.z = ((float)(z / 256.0) + aabb_size) * (scale->max.z - scale->min.z) - scale->min.z;
}

__global__ void create_octree(
  const octree_generation_position *const sorted_positions,
  const size_t *const nodes_difference,
  size_t nb_objects,
  const struct AABB *const scale,
  struct octree *resulting_octree)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= nb_objects) return; // Nothing to do here

  /*
   * As the objects are sorted, only two things must be done:
   * - if the object create a new hierachy
   *    - Set the center of the octree
   *    - set the start range
   *    - Set the parent pointer of the children
   * - if the object end the hierachy (next one create a new hierachy)
   *    - Set the end range
   */


  size_t previous_diff = (index == 0 ? 0 : nodes_difference[index - 1]);
  size_t current_diff = nodes_difference[index];

  if (previous_diff != current_diff)
  {// This object create a new hierachy

    // Get the center of the octree.
    // To do that, perform the inverse trick of get_point_position
    // to get a value in the [0-1[ range, and scale it back to get the center.
    octree_generation_position current_position = sorted_positions[index];
    uint8_t x, y, z, level = get_level(current_position);
    for (uint8_t i = 0; i < 8; ++i)
    {
      uint8_t local_position = get_level_position(current_position, i + 1);
      x = x << 1 | ((local_position & 1) != 0);
      y = y << 1 | ((local_position & 2) != 0);
      z = z << 1 | ((local_position & 4) != 0);
    }

    // The new node have been created, but unused
    // so their start and end must be set.
    for (size_t i = previous_diff + 1/* Skip the already created parent */; i < current_diff; ++i)
    {
      resulting_octree[i - 1].start_index = index;
      resulting_octree[i - 1].end_index = index;
      get_aabb_box(x, y, z, level - (current_diff - i), scale, &resulting_octree[i - 1].box);
    }

    // Set the starting index.
    resulting_octree[current_diff - 1].start_index = index;
    get_aabb_box(x, y, z, level, scale, & resulting_octree[current_diff - 1].box);
  }

  if (index + 1 >= nb_objects || nodes_difference[index + 1] != current_diff)
  {// This object end the current hierachy
    resulting_octree[current_diff - 1].end_index = index + 1;
  }

  // As the parent finding research might need to iterate over
  // all previous objects, do it only when all start and end range
  // have been done, so that most object can safely be skipped over.
  __syncthreads();


  if (previous_diff != current_diff)
  {// Create a new hierachy, find parents and set the children
    octree_generation_position current_position = sorted_positions[index];
    size_t bottom_level = get_level(current_position);

    // First, fixes all created nodes, except the last (there is nothing to fix here)
    for (size_t index = previous_diff; index + 1 < current_diff; ++index)
    {
      uint8_t fix_level = bottom_level - (current_diff - (index + 2));
      resulting_octree[index].children[
        get_level_position(current_position, fix_level)
      ] = (resulting_octree + index + 1);
    }

    size_t fix_level = get_level(current_position) - (current_diff - previous_diff);

    // Then fix to highest created node as it's parent is not known.
    // First find it's parent.
    if (previous_diff != 0)
    {// Can't fix the root node
      size_t parent_octree_index = previous_diff - 1;

      while (parent_octree_index > 0)
      {
        size_t parent_first_object_index = resulting_octree[parent_octree_index].start_index;
        octree_generation_position parent_position = sorted_positions[parent_first_object_index];
        size_t parent_level = get_level(parent_position);

        // Climb up the parent chain
        while (parent_octree_index > 0 && fix_level < parent_level &&
               resulting_octree[parent_octree_index - 1].start_index == parent_first_object_index)
        {
          --parent_octree_index;
          --parent_level;
        }

        if (parent_level == fix_level)
        {
          break;
        }

        --parent_octree_index;
      }

      // Write the children to it's position
      resulting_octree[parent_octree_index].children[
          get_level_position(current_position, fix_level + 1)
      ] = (resulting_octree + previous_diff);
    }
  }
}

void create_octree(
  struct scene *scene,
  struct octree **octree)
{
  struct scene CPU_scene;
  hipMemcpy(&CPU_scene, scene, sizeof(struct scene), hipMemcpyDefault);

  dim3 threadsPerBlock(32);
  dim3 numBlocks(ceil(CPU_scene.object_count * 1.0 / threadsPerBlock.x));

  // Compute the bounding box
  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * CPU_scene.object_count);
  object_compute_bounding_box<<<numBlocks, threadsPerBlock>>>(scene, aabbs);

  // Compute the global scale
  struct AABB *resulting_scale;
  hipMalloc(&resulting_scale, sizeof(struct AABB));
  find_scene_scale_shared<<<numBlocks, threadsPerBlock>>>(aabbs, CPU_scene.object_count, resulting_scale);

  // Compute the position of the objects
  octree_generation_position *positions;
  hipMalloc(&positions, sizeof(octree_generation_position) * CPU_scene.object_count);
  position_object<<<numBlocks, threadsPerBlock>>>(aabbs, resulting_scale, positions, CPU_scene.object_count);

  // Sort the position of the objects
  parallel_radix_sort(positions, CPU_scene.objects, CPU_scene.object_count);

  // Get the number of nodes needed per each objects
  size_t *node_differences;
  hipMalloc(&node_differences, sizeof(size_t) * CPU_scene.object_count);
  nodes_difference_array<<<numBlocks, threadsPerBlock>>>(positions, node_differences, CPU_scene.object_count);

  // Perform a prefix sum on it
  shared_prefix_sum(node_differences, CPU_scene.object_count);

  // Create the resulting octree
  size_t nb_nodes;
  hipMemcpy(&nb_nodes, node_differences + (CPU_scene.object_count - 1), sizeof(size_t), hipMemcpyDefault);

  hipMalloc(octree, sizeof(struct octree) * nb_nodes);
  hipMemset(*octree, 0, sizeof(struct octree) * nb_nodes);
  create_octree<<<numBlocks, threadsPerBlock>>>(positions, node_differences, CPU_scene.object_count, resulting_scale, *octree);

  hipFree(aabbs);
  hipFree(resulting_scale);
  hipFree(positions);
  hipFree(node_differences);
}