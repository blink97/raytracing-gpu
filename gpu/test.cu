#include "hip/hip_runtime.h"
#include <err.h>

#include "parser.h"
#include "partitioning/aabb.h"
#include "partitioning/octree.h"

#include <iostream>
#include <bitset>

#define TESTS_PATH "../../tests/"

#define CUBE TESTS_PATH "cube.svati"
#define ISLAND_SMOOTH TESTS_PATH "island_smooth.svati" // High objects count
#define DARK_NIGHT TESTS_PATH "dark-night.svati" // Second highest objects count
#define SPHERES TESTS_PATH "spheres.svati"

/*
 * File used for test purpose,
 * to see if the cuda functions have the correct output.
 */

void display_last_error()
{
  auto err = hipGetLastError();
  if (err != hipSuccess)
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
}


void display_GPU_memory()
{
  display_last_error();
  size_t free, total;
  hipMemGetInfo(&free, &total);

  std::cout << "memory: " << free << "/" << total << std::endl;
}

void display_cuda_scene(const struct scene *cuda_scene)
{
  std::cout << "cuda scene:" << std::endl;
  struct scene CPU_scene;
  hipMemcpy(&CPU_scene, cuda_scene, sizeof(struct scene), hipMemcpyDefault);

  std::cout << "camera size: " << CPU_scene.camera.width << " " << CPU_scene.camera.height << std::endl;

  struct object *objects = new struct object[CPU_scene.object_count];
  hipMemcpy(objects, CPU_scene.objects, sizeof(struct object) * CPU_scene.object_count, hipMemcpyDefault);

  vector3 vertex[3];
  vector3 normal[3];

  for (size_t i = 0; i < CPU_scene.object_count; ++i)
  {
    std::cout << "new object with " << objects[i].triangle_count << " triangles" << std::endl;

    for (int j = 0; j < objects[i].triangle_count; ++j)
    {
      hipMemcpy(vertex, get_vertex(objects[i].triangles, j), sizeof(vector3) * 3, hipMemcpyDefault);
      hipMemcpy(normal, get_normal(objects[i].triangles, j), sizeof(vector3) * 3, hipMemcpyDefault);

      std::cout << "v : "
                << vertex[0].x << "," << vertex[0].y << "," << vertex[0].z << "\t| "
                << vertex[1].x << "," << vertex[1].y << "," << vertex[1].z << "\t| "
                << vertex[2].x << "," << vertex[2].y << "," << vertex[2].z << std::endl;
      std::cout << "vn: "
                << normal[0].x << "," << normal[0].y << "," << normal[0].z << "\t| "
                << normal[1].x << "," << normal[1].y << "," << normal[1].z << "\t| "
                << normal[2].x << "," << normal[2].y << "," << normal[2].z << std::endl;
    }
  }

  delete[] objects;
}

void display_aabbs(const struct AABB *aabbs, size_t nb_objects)
{
  std::cout << "displaying aabb" << std::endl;

  struct AABB *cpu_aabbs;
  hipHostMalloc(&cpu_aabbs, sizeof(struct AABB) * nb_objects);
  hipMemcpy(cpu_aabbs, aabbs, sizeof(struct AABB) * nb_objects, hipMemcpyDefault);

  std::cout << std::endl << nb_objects << " objects AABB (from: " << aabbs << " to: " << cpu_aabbs << ")" << std::endl;

  for (int i = 0; i < nb_objects; ++i)
  {//Display the aabb
    struct AABB current = cpu_aabbs[i];

    std::cout << current.min.x << "," << current.min.y << "," << current.min.z << " - "
              << current.max.x << "," << current.max.y << "," << current.max.z << std::endl;
  }

  hipHostFree(cpu_aabbs);
}

void display_positions(
  const octree_generation_position *positions,
  size_t *positions_sorted_index,
  size_t nb_objects)
{
  std::cout << "displaying positions" << (positions_sorted_index ? " sorted": " unsorted") << std::endl;

  octree_generation_position *cpu_positions;
  hipHostMalloc(&cpu_positions, sizeof(octree_generation_position) * nb_objects);
  hipMemcpy(cpu_positions, positions, sizeof(octree_generation_position) * nb_objects, hipMemcpyDefault);

  size_t *cpu_positions_sorted_index = nullptr;
  if (positions_sorted_index)
  {
    hipHostMalloc(&cpu_positions_sorted_index, sizeof(size_t) * nb_objects);
    hipMemcpy(cpu_positions_sorted_index, positions_sorted_index, sizeof(size_t) * nb_objects, hipMemcpyDefault);
  }

  for (int i = 0; i < nb_objects; ++i)
  {
    std::cout << "level: " << ((cpu_positions[i] & 0xFF000000) >> 24)
              << " " << std::bitset<24>(cpu_positions[i]);

    if (positions_sorted_index)
    {
      std::cout << " position: " << cpu_positions_sorted_index[i];
    }

    std::cout << std::endl;
  }

  hipHostFree(cpu_positions);
  if (positions_sorted_index) hipHostFree(cpu_positions_sorted_index);
}

void display_node_differences(
  const octree_generation_position *positions,
  size_t *node_differences,
  size_t nb_objects)
{
  std::cout << "displaying nodes differences" << std::endl;

  octree_generation_position *cpu_positions;
  hipHostMalloc(&cpu_positions, sizeof(octree_generation_position) * nb_objects);
  hipMemcpy(cpu_positions, positions, sizeof(octree_generation_position) * nb_objects, hipMemcpyDefault);

  size_t *cpu_node_differences;
  hipHostMalloc(&cpu_node_differences, sizeof(size_t) * nb_objects);
  hipMemcpy(cpu_node_differences, node_differences, sizeof(size_t) * nb_objects, hipMemcpyDefault);

  for (int i = 0; i < nb_objects; ++i)
  {
    std::cout << "level: " << ((cpu_positions[i] & 0xFF000000) >> 24)
              << " " << std::bitset<24>(cpu_positions[i])
              << " diff: " << cpu_node_differences[i] << std::endl;
  }

  hipHostFree(cpu_positions);
  hipHostFree(cpu_node_differences);
}

void test_partitioning(const struct scene *cuda_scene)
{
  display_cuda_scene(cuda_scene);

  struct scene CPU_scene;
  hipMemcpy(&CPU_scene, cuda_scene, sizeof(struct scene), hipMemcpyDefault);

  dim3 threadsPerBlock(32);
  dim3 numBlocks(ceil(CPU_scene.object_count * 1.0 / threadsPerBlock.x));

  std::cout << "kernel param: " << numBlocks.x << " " << threadsPerBlock.x << std::endl;
  std::cout << "nb_objects: " << CPU_scene.object_count << std::endl;

  // Compute the bounding box
  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * CPU_scene.object_count);
  object_compute_bounding_box<<<numBlocks, threadsPerBlock>>>(cuda_scene, aabbs);
  display_aabbs(aabbs, CPU_scene.object_count);

  // Compute the global scale
  struct AABB *resulting_scale;
  hipMalloc(&resulting_scale, sizeof(struct AABB));
  find_scene_scale_shared<<<numBlocks, threadsPerBlock>>>(aabbs, CPU_scene.object_count, resulting_scale);
  display_aabbs(resulting_scale, 1);

  // Compute the position of the objects
  octree_generation_position *positions;
  hipMalloc(&positions, sizeof(octree_generation_position) * CPU_scene.object_count);
  position_object<<<numBlocks, threadsPerBlock>>>(aabbs, resulting_scale, positions, CPU_scene.object_count);
  display_positions(positions, nullptr, CPU_scene.object_count);

  // Sort the position of the objects
  size_t *positions_sorted_index;
  hipMalloc(&positions_sorted_index, sizeof(size_t) * CPU_scene.object_count);
  single_thread_bubble_argsort<<<1, 1>>>(positions, positions_sorted_index, CPU_scene.object_count);
  display_positions(positions, positions_sorted_index, CPU_scene.object_count);

  // Get the number of nodes needed per each objects
  size_t *node_differences;
  hipMalloc(&node_differences, sizeof(size_t) * CPU_scene.object_count);
  nodes_difference_array<<<numBlocks, threadsPerBlock>>>(positions, node_differences, CPU_scene.object_count);
  display_node_differences(positions, node_differences, CPU_scene.object_count);

  // Perform a prefix sum on it
  single_thread_nodes_difference_to_prefix_array<<<1, 1>>>(node_differences, CPU_scene.object_count);
  display_node_differences(positions, node_differences, CPU_scene.object_count);

  hipFree(aabbs);
  hipFree(resulting_scale);
  hipFree(positions);
  hipFree(node_differences);
}

int main(int argc, char *argv[])
{

#  if defined(LAYOUT_FRAGMENTED)
  std::cout << "Using fragmented layout" << std::endl;
#  elif defined(LAYOUT_AOS)
  std::cout << "Using array of structures (AOS) layout" << std::endl;
#  else /* LAYOUT_SOA */
  std::cout << "Using structure of arrays (SOA) layout" << std::endl;
#  endif

  display_GPU_memory();

  //struct scene scene = parser(CUBE);
  //struct scene scene = parser(DARK_NIGHT);
  struct scene scene = parser(ISLAND_SMOOTH);
  //struct scene scene = parser(SPHERES);

  display_GPU_memory();

  struct scene *cuda_scene = to_cuda(&scene);

  test_partitioning(cuda_scene);
}
