#include "hip/hip_runtime.h"
#include <err.h>

#include "parser.h"
#include "partitioning/aabb.h"
#include "partitioning/octree.h"
#include "partitioning/prefix_sum.h"
#include "partitioning/sort.h"
#include "partitioning/utils.h"

#include <cassert>
#include <iostream>
#include <string>
#include <bitset>

#define TESTS_PATH "../../tests/"

#define CUBE TESTS_PATH "cube.svati"
#define ISLAND_SMOOTH TESTS_PATH "island_smooth.svati" // High objects count
#define DARK_NIGHT TESTS_PATH "dark-night.svati" // Second highest objects count
#define SPHERES TESTS_PATH "spheres.svati"

/*
 * File used for test purpose,
 * to see if the cuda functions have the correct output.
 */

void display_last_error()
{
  auto err = hipGetLastError();
  if (err != hipSuccess)
    std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
}


void display_GPU_memory()
{
  display_last_error();
  size_t free, total;
  hipMemGetInfo(&free, &total);

  std::cout << "memory: " << free << "/" << total << std::endl;
}

void display_cuda_scene(const struct scene *cuda_scene)
{
  std::cout << "cuda scene:" << std::endl;
  struct scene CPU_scene;
  hipMemcpy(&CPU_scene, cuda_scene, sizeof(struct scene), hipMemcpyDefault);

  std::cout << "camera size: " << CPU_scene.camera.width << " " << CPU_scene.camera.height << std::endl;

  struct object *objects = new struct object[CPU_scene.object_count];
  hipMemcpy(objects, CPU_scene.objects, sizeof(struct object) * CPU_scene.object_count, hipMemcpyDefault);

  vector3 vertex[3];
  vector3 normal[3];

  for (size_t i = 0; i < CPU_scene.object_count; ++i)
  {
    std::cout << "new object with " << objects[i].triangle_count << " triangles" << std::endl;

    for (int j = 0; j < objects[i].triangle_count; ++j)
    {
      hipMemcpy(vertex, get_vertex(objects[i].triangles, j), sizeof(vector3) * 3, hipMemcpyDefault);
      hipMemcpy(normal, get_normal(objects[i].triangles, j), sizeof(vector3) * 3, hipMemcpyDefault);

      std::cout << "v : "
                << vertex[0].x << "," << vertex[0].y << "," << vertex[0].z << "\t| "
                << vertex[1].x << "," << vertex[1].y << "," << vertex[1].z << "\t| "
                << vertex[2].x << "," << vertex[2].y << "," << vertex[2].z << std::endl;
      std::cout << "vn: "
                << normal[0].x << "," << normal[0].y << "," << normal[0].z << "\t| "
                << normal[1].x << "," << normal[1].y << "," << normal[1].z << "\t| "
                << normal[2].x << "," << normal[2].y << "," << normal[2].z << std::endl;
    }
  }

  delete[] objects;
}

void display_aabbs(const struct AABB *aabbs, size_t nb_objects)
{
  std::cout << "displaying aabb" << std::endl;

  struct AABB *cpu_aabbs;
  hipHostMalloc(&cpu_aabbs, sizeof(struct AABB) * nb_objects);
  hipMemcpy(cpu_aabbs, aabbs, sizeof(struct AABB) * nb_objects, hipMemcpyDefault);

  std::cout << std::endl << nb_objects << " objects AABB (from: " << aabbs << " to: " << cpu_aabbs << ")" << std::endl;

  for (int i = 0; i < nb_objects; ++i)
  {//Display the aabb
    struct AABB current = cpu_aabbs[i];

    std::cout << current.min.x << "," << current.min.y << "," << current.min.z << " || "
              << current.max.x << "," << current.max.y << "," << current.max.z << std::endl;
  }

  hipHostFree(cpu_aabbs);
}

void display_positions(
  const octree_generation_position *positions,
  struct object *objects,
  size_t nb_objects)
{
  std::cout << "displaying positions" << (objects ? " sorted": " unsorted") << std::endl;

  octree_generation_position *cpu_positions;
  hipHostMalloc(&cpu_positions, sizeof(octree_generation_position) * nb_objects);
  hipMemcpy(cpu_positions, positions, sizeof(octree_generation_position) * nb_objects, hipMemcpyDefault);

  struct object *cpu_objects = nullptr;
  if (objects)
  {
    hipHostMalloc(&cpu_objects, sizeof(struct object) * nb_objects);
    hipMemcpy(cpu_objects, objects, sizeof(struct object) * nb_objects, hipMemcpyDefault);
  }

  for (int i = 0; i < nb_objects; ++i)
  {
    std::cout << "level: " << (int)get_level(cpu_positions[i])
              << " " << std::bitset<32>(cpu_positions[i]);

    if (cpu_objects)
    {
      std::cout << " triangle count: " << cpu_objects[i].triangle_count;
    }

    std::cout << std::endl;
  }

  hipHostFree(cpu_positions);
  if (cpu_objects)
    hipHostFree(cpu_objects);
}

void display_node_differences(
  const octree_generation_position *positions,
  size_t *node_differences,
  size_t nb_objects)
{
  std::cout << "displaying nodes differences" << std::endl;

  octree_generation_position *cpu_positions;
  hipHostMalloc(&cpu_positions, sizeof(octree_generation_position) * nb_objects);
  hipMemcpy(cpu_positions, positions, sizeof(octree_generation_position) * nb_objects, hipMemcpyDefault);

  size_t *cpu_node_differences;
  hipHostMalloc(&cpu_node_differences, sizeof(size_t) * nb_objects);
  hipMemcpy(cpu_node_differences, node_differences, sizeof(size_t) * nb_objects, hipMemcpyDefault);

  for (int i = 0; i < nb_objects; ++i)
  {
    std::cout << "level: " << (int)get_level(cpu_positions[i])
              << " " << std::bitset<32>(cpu_positions[i])
              << " diff: " << cpu_node_differences[i] << std::endl;
  }

  hipHostFree(cpu_positions);
  hipHostFree(cpu_node_differences);
}

void display_octree_iter(
  const struct octree *const octree,
  const octree_generation_position *const positions,
  size_t nb_nodes)
{
  struct octree *cpu_octree;
  hipHostMalloc(&cpu_octree, sizeof(struct octree) * nb_nodes);
  hipMemcpy(cpu_octree, octree, sizeof(struct octree) * nb_nodes, hipMemcpyDefault);

  for (size_t i = 0; i < nb_nodes; ++i)
  {
    octree_generation_position start_pos;
    hipMemcpy(&start_pos, positions + cpu_octree[i].start_index, sizeof(octree_generation_position), hipMemcpyDefault);

    std::cout << "index: " << i << std::endl
              << "box: " << cpu_octree[i].box.min.x << "," << cpu_octree[i].box.min.y << "," << cpu_octree[i].box.min.z
              << " || " << cpu_octree[i].box.max.x << "," << cpu_octree[i].box.max.y << "," << cpu_octree[i].box.max.z << std::endl
              << "range (" << cpu_octree[i].start_index << "," << cpu_octree[i].end_index << ")" << std::endl
              << "level: " << (int)get_level(start_pos) << std::endl
              << "position: " << std::bitset<32>(start_pos) << std::endl;

    for (int j = 0; j < 8; ++j)
    {
      std::cout << "children: " << std::bitset<3>(j) << " (";
      if (cpu_octree[i].children[j])
        std::cout << (cpu_octree[i].children[j] - octree);
      else
        std::cout << "nullptr";

      std::cout << ")" << std::endl;
    }

    std::cout << std::endl;
  }

  hipHostFree(cpu_octree);
}

void display_octree_rec(const struct octree *const octree, size_t current_level = 0)
{
  struct octree cpu_octree;
  hipMemcpy(&cpu_octree, octree, sizeof(struct octree), hipMemcpyDefault);

  auto indent = std::string(current_level, '\t');

  std::cout << indent << "box: " << cpu_octree.box.min.x << "," << cpu_octree.box.min.y << "," << cpu_octree.box.min.z
            << " || " << cpu_octree.box.max.x << "," << cpu_octree.box.max.y << "," << cpu_octree.box.max.z << std::endl
            << indent << "range (" << cpu_octree.start_index << "," << cpu_octree.end_index << ")" << std::endl;

  for (int i = 0; i < 8; ++i)
  {
    std::cout << indent << "children: " << std::bitset<3>(i) << std::endl;
    if (cpu_octree.children[i])
      display_octree_rec(cpu_octree.children[i], current_level + 1);
  }
}

void test_partitioning(const struct scene *cuda_scene)
{
  //display_cuda_scene(cuda_scene);

  struct scene CPU_scene;
  hipMemcpy(&CPU_scene, cuda_scene, sizeof(struct scene), hipMemcpyDefault);

  dim3 threadsPerBlock(32);
  dim3 numBlocks(ceil(CPU_scene.object_count * 1.0 / threadsPerBlock.x));

  std::cout << "kernel param: " << numBlocks.x << " " << threadsPerBlock.x << std::endl;
  std::cout << "nb_objects: " << CPU_scene.object_count << std::endl;

  // Compute the bounding box
  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * CPU_scene.object_count);
  compute_bounding_box(cuda_scene, aabbs);
  display_aabbs(aabbs, CPU_scene.object_count);

  // Compute the global scale
  struct AABB *resulting_scale;
  hipMalloc(&resulting_scale, sizeof(struct AABB));
  //find_scene_scale_basic<<<numBlocks, threadsPerBlock>>>(aabbs, CPU_scene.object_count, resulting_scale);
  find_scene_scale_shared<<<numBlocks, threadsPerBlock>>>(aabbs, CPU_scene.object_count, resulting_scale);
  display_aabbs(resulting_scale, 1);

  // Compute the position of the objects
  octree_generation_position *positions;
  hipMalloc(&positions, sizeof(octree_generation_position) * CPU_scene.object_count);
  position_object<<<numBlocks, threadsPerBlock>>>(aabbs, resulting_scale, positions, CPU_scene.object_count);
  display_positions(positions, nullptr, CPU_scene.object_count);

  // Sort the position of the objects
  //single_thread_bubble_sort(positions, CPU_scene.objects, CPU_scene.object_count);
  parallel_radix_sort(positions, CPU_scene.objects, CPU_scene.object_count);
  display_positions(positions, CPU_scene.objects, CPU_scene.object_count);

  // Get the number of nodes needed per each objects
  size_t *node_differences;
  hipMalloc(&node_differences, sizeof(size_t) * CPU_scene.object_count);
  nodes_difference_array<<<numBlocks, threadsPerBlock>>>(positions, node_differences, CPU_scene.object_count);
  display_node_differences(positions, node_differences, CPU_scene.object_count);

  // Perform a prefix sum on it
  shared_prefix_sum(node_differences, CPU_scene.object_count);
  display_node_differences(positions, node_differences, CPU_scene.object_count);

  // Create the resulting octree
  size_t nb_nodes;
  hipMemcpy(&nb_nodes, node_differences + (CPU_scene.object_count - 1), sizeof(size_t), hipMemcpyDefault);

  std::cout << "nb nodes in the resulting octree: "<< nb_nodes << std::endl;

  struct octree *octree;
  hipMalloc(&octree, sizeof(struct octree) * nb_nodes);
  hipMemset(octree, 0, sizeof(struct octree) * nb_nodes);
  create_octree<<<numBlocks, threadsPerBlock>>>(positions, node_differences, CPU_scene.object_count, resulting_scale, octree);
  display_octree_iter(octree, positions, nb_nodes);
  display_octree_rec(octree);

  hipFree(octree);
  hipFree(aabbs);
  hipFree(resulting_scale);
  hipFree(positions);
  hipFree(node_differences);
}

void test_prefix_sum()
{
  constexpr size_t size = 15;
  size_t cpu_values[size];
  for (size_t i = 0; i < size; ++i)
    cpu_values[i] = i + 1;

  size_t *values;
  hipMalloc(&values, sizeof(size_t) * size);
  hipMemcpy(values, cpu_values, sizeof(size_t) * size, hipMemcpyDefault);
  shared_prefix_sum(values, size);
  hipMemcpy(cpu_values, values, sizeof(size_t) * size, hipMemcpyDefault);

  for (size_t i = 0; i < size; ++i)
    std::cout << (i + 1) << ": " << cpu_values[i] << std::endl;

  hipFree(values);
}


void test_sort()
{
  constexpr size_t size = 385;
  uint32_t *array = new uint32_t[size];

  // Random initialisation
  for (size_t i = 0; i < size; ++i)
    array[i] = i;

  uint32_t *GPU_keys;
  hipMalloc(&GPU_keys, sizeof(uint32_t) * size);

  // Is absolutely not used, but is needed for the function
  size_t *GPU_values;
  hipMalloc(&GPU_values, sizeof(size_t) * size);

  hipMemcpy(GPU_keys, array, sizeof(uint32_t) * size, hipMemcpyDefault);
  parallel_radix_sort(GPU_keys, GPU_values, size);

  // Assert that the values are sorted
  hipMemcpy(array, GPU_keys, sizeof(uint32_t) * size, hipMemcpyDefault);
  for (size_t i = 0; i + 1 < size; ++i)
  {
    std::cout << i << ": " << array[i] << " " << array[i + 1]
              << " bits: " << std::bitset<32>(array[i]) << " " << std::bitset<32>(array[i + 1]) << std::endl;
  }

  delete[] array;
  hipFree(GPU_keys);
  hipFree(GPU_values);
}


void test_octree_creation(struct scene *cuda_scene)
{
  struct octree *octree;
  create_octree(cuda_scene, &octree);

  display_octree_rec(octree);
}


void test_aabb_hit()
{
  struct AABB scale = {
    .min = { .x = 0, .y = 0, .z = 0 },
    .max = { .x = 1, .y = 1, .z = 1 },
  };

  struct ray ray {
    .origin = { .x = 0.0, .y = 0.0, .z = 0.0 },
    .direction = { .x = 0.0, .y = 0.0, .z = 1.0 }
  };
  // Test ray intersection.
  std::cout << "ray it aabb: " << hit_aabb(&scale, &ray) << std::endl;
}

int main(int argc, char *argv[])
{

#  if defined(LAYOUT_FRAGMENTED)
  std::cout << "Using fragmented layout" << std::endl;
#  elif defined(LAYOUT_AOS)
  std::cout << "Using array of structures (AOS) layout" << std::endl;
#  else /* LAYOUT_SOA */
  std::cout << "Using structure of arrays (SOA) layout" << std::endl;
#  endif

  display_GPU_memory();

  //struct scene scene = parser(CUBE);
  //struct scene scene = parser(DARK_NIGHT);
  //struct scene scene = parser(ISLAND_SMOOTH);
  struct scene scene = parser(SPHERES);

  display_GPU_memory();

  struct scene *cuda_scene = to_cuda(&scene);

  test_partitioning(cuda_scene);
  //test_prefix_sum();
  //test_sort();
  //test_octree_creation(cuda_scene);
  //test_aabb_hit();
}