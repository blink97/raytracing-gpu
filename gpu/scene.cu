#include "scene.h"

# include <cstdlib>

#include "partitioning/aabb.h"
#include "partitioning/octree.h"

/* Layout dependent code */
# if defined(LAYOUT_FRAGMENTED)

__host__ __device__ vector3 *get_vertex(const struct triangles_layout triangles, uint32_t triangle_index)
{
  return triangles.data + triangle_index * 6;
}

__host__ __device__ vector3 *get_normal(const struct triangles_layout triangles, uint32_t triangle_index)
{
  return triangles.data + triangle_index * 6 + 3/* Skip the vertex part */;
}


# elif defined(LAYOUT_AOS)


__host__ __device__ vector3 *get_vertex(const struct triangles_layout triangles, uint32_t triangle_index)
{
  return triangles.data + triangle_index * 6;
}

__host__ __device__ vector3 *get_normal(const struct triangles_layout triangles, uint32_t triangle_index)
{
  return triangles.data + triangle_index * 6 + 3/* Skip the vertex part */;
}


# else /* LAYOUT_SOA */


__host__ __device__ vector3 *get_vertex(const struct triangles_layout triangles, uint32_t triangle_index)
{
  return triangles.vertex + triangle_index * 3;
}

__host__ __device__ vector3 *get_normal(const struct triangles_layout triangles, uint32_t triangle_index)
{
  return triangles.normal + triangle_index * 3;
}


# endif
/* End of layout dependent code */


/**
 * Rewrite the triangles pointers so that they point to the correct position
 */
static void rewrite_pointers(const struct scene *scene)
{
  constexpr uint32_t objects_block_size = 128;

  size_t offset = 0;
  for (uint32_t current = 0; current < scene->object_count; current += objects_block_size)
  {
    // Get the object back on the CPU
    struct object current_objects[objects_block_size];

    uint32_t current_size = ((scene->object_count <= current + objects_block_size)
        ? scene->object_count - current
        : objects_block_size
    );

    hipMemcpy(
      &current_objects,
      scene->objects + current,
      sizeof(struct object) * current_size,
      hipMemcpyDefault
    );

    for (uint32_t i = 0; i < current_size; ++i)
    {

      /* Layout dependent code */
      # if defined(LAYOUT_FRAGMENTED)

        /* Nothing to do here */

      # elif defined(LAYOUT_AOS)

        current_objects[i].triangles.data = scene->objects_data.vertex_and_normal + offset;

        offset += 6 * current_object.triangle_count;

      # else /* LAYOUT_SOA */

        current_objects[i].triangles.vertex = scene->objects_data.vertex + offset;
        current_objects[i].triangles.normal = scene->objects_data.normal + offset;

        offset += 3 * current_objects[i].triangle_count;

      # endif
      /* End of layout dependent code */
    }


    // Replace the object at it's current location
    hipMemcpy(
      scene->objects + current,
      &current_objects,
      sizeof(struct object) * current_size,
      hipMemcpyDefault
    );
  }
}


struct scene empty_scene()
{
  struct scene scene = {
    .objects_data = {/* Default initialisation */},
    .objects = nullptr,
    .object_count = 0,
    .lights = nullptr,
    .light_count = 0,
    .camera = {/* Use default init */},
    .triangle_count = 0,

    /* Partitioning dependent code */
# if defined(PARTITIONING_AABB) || defined(PARTITIONING_OCTREE)
    .aabbs = nullptr,
#  if defined(PARTITIONING_OCTREE)
    .octree = nullptr,
#  endif
# endif
/* End of Partitioning dependent code */

  };

  return scene;
}


/*
 * Add another object with the given triangle count to the scene.
 * The object is returned so that vertex and normal can be set.
 */
struct object *add_object_to_scene(struct scene *scene, uint32_t nb_triangles)
{
  scene->objects = (struct object *)realloc(scene->objects, sizeof(struct object) * (scene->object_count + 1));

  /* Layout dependent code */
#  if defined(LAYOUT_FRAGMENTED)

  struct triangles_layout triangles = {
    .data = (vector3 *) malloc(sizeof(vector3) * 6/* 3 vertex and 3 normal */ * nb_triangles)
  };

#  elif defined(LAYOUT_AOS)

  // Extend the global triangles storage
  vector3 *old_ptr = scene->objects_data.vertex_and_normal;

  scene->objects_data.vertex_and_normal = (vector3 *)realloc(
    scene->objects_data.vertex_and_normal,
    sizeof(vector3) * 6/* 3 vertex and 3 normal */ * (scene->triangle_count + nb_triangles)
  );

  if (old_ptr != scene->objects_data.vertex_and_normal)
  {// Rewrite the pointers as it may have changed.
    rewrite_pointers(scene);
  }

  struct triangles_layout triangles = {
    .data = &scene->objects_data.vertex_and_normal[scene->triangle_count * 6]
  };

#  else /* LAYOUT_SOA */

  vector3 *old_vertex_ptr = scene->objects_data.vertex;
  vector3 *old_normal_ptr = scene->objects_data.normal;

  // Extend the global vertex and normal storage
  scene->objects_data.vertex = (vector3 *)realloc(scene->objects_data.vertex, sizeof(vector3) * 3 * (scene->triangle_count + nb_triangles));
  scene->objects_data.normal = (vector3 *)realloc(scene->objects_data.normal, sizeof(vector3) * 3 * (scene->triangle_count + nb_triangles));

  if (old_vertex_ptr != scene->objects_data.vertex || old_normal_ptr != scene->objects_data.normal)
  {// Rewrite the pointers as it may have changed.
    rewrite_pointers(scene);
  }

  struct triangles_layout triangles = {
    .vertex = &scene->objects_data.vertex[scene->triangle_count * 3],
    .normal = &scene->objects_data.normal[scene->triangle_count * 3]
  };

#  endif
  /* End of layout dependent code */

  // Create a default objects
  struct object new_object = {
    .triangles = triangles,
    .triangle_count = nb_triangles,
    .ka = { .x = 0, .y = 0, .z = 0 },
    .kd = { .x = 0, .y = 0, .z = 0 },
    .ks = { .x = 0, .y = 0, .z = 0 },
    .ns = 0,
    .ni = 1,
    .nr = 0,
    .d = 1
  };

  struct object *current_object = &scene->objects[scene->object_count];
  *current_object = new_object;

  scene->object_count++;
  scene->triangle_count += nb_triangles;

  return current_object;
}

/*
 * Get a new scene, containing only cuda memory so that
 * it can be used in a GPU context.
 */
struct scene *to_cuda(const struct scene *const scene)
{
  struct scene cuda_scene = {
    .objects_data = scene->objects_data,
    .objects = nullptr,
    .object_count = scene->object_count,
    .lights = nullptr,
    .light_count = scene->light_count,
    .camera = scene->camera,
    .triangle_count = scene->triangle_count,
  };

  hipMalloc(&cuda_scene.objects, sizeof(struct object) * cuda_scene.object_count);
  hipMalloc(&cuda_scene.lights, sizeof(struct light) * cuda_scene.light_count);

  hipMemcpy(cuda_scene.objects, scene->objects, sizeof(struct object) * cuda_scene.object_count, hipMemcpyHostToDevice);
  hipMemcpy(cuda_scene.lights, scene->lights, sizeof(struct light) * cuda_scene.light_count, hipMemcpyHostToDevice);

  /* Layout dependent code */
#  if defined(LAYOUT_FRAGMENTED)

  // Copy all vertex and normal arrays in each objects.
  for (uint32_t i = 0; i < scene->object_count; ++i)
  {
    size_t mem_size = sizeof(vector3) * 6 /* 3 vertex and 3 normal */ * scene->objects[i].triangle_count;

    struct object current_object = scene->objects[i];
    hipMalloc(&current_object.triangles.data, mem_size);

    // Copy the triangle
    hipMemcpy(
      current_object.triangles.data,
      scene->objects[i].triangles.data,
      mem_size,
      hipMemcpyHostToDevice
    );

    // Copy the object back to GPU
    hipMemcpy(
      &cuda_scene.objects[i],
      &current_object,
      sizeof(struct object),
      hipMemcpyHostToDevice
    );
  }

#  elif defined(LAYOUT_AOS)

  // Copy the global vertex and normal array,
  // and rewrite the objects triangles pointers.

  size_t mem_size = sizeof(vector3) * 6 * scene->triangle_count;

  hipMalloc(&cuda_scene.objects_data.vertex_and_normal, mem_size);
  hipMemcpy(
    cuda_scene.objects_data.vertex_and_normal,
    scene->objects_data.vertex_and_normal,
    mem_size,
    hipMemcpyHostToDevice
  );

  // Rewrite the pointers so that they point to the correct value.
  rewrite_pointers(&cuda_scene);

#  else /* LAYOUT_SOA */

  // Copy the globals vertex and normal arrays,
  // and rewrite the objects triangles pointers.

  size_t mem_size = sizeof(vector3) * 3 * scene->triangle_count;

  hipMalloc(&cuda_scene.objects_data.vertex, mem_size);
  hipMalloc(&cuda_scene.objects_data.normal, mem_size);
  hipMemcpy(
    cuda_scene.objects_data.vertex,
    scene->objects_data.vertex,
    mem_size,
    hipMemcpyHostToDevice
  );
  hipMemcpy(
    cuda_scene.objects_data.normal,
    scene->objects_data.normal,
    mem_size,
    hipMemcpyHostToDevice
  );

  // Rewrite the pointers so that they point to the correct value.
  rewrite_pointers(&cuda_scene);

#  endif

  // Copy the FULL scene to GPU memory
  struct scene *GPU_cuda_scene;
  hipMalloc(&GPU_cuda_scene, sizeof(struct scene));
  hipMemcpy(GPU_cuda_scene, &cuda_scene, sizeof(struct scene), hipMemcpyDefault);

/* Partitioning dependent code */
# if defined(PARTITIONING_AABB)

  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * cuda_scene.object_count);

  compute_bounding_box(GPU_cuda_scene, aabbs);

  hipMemcpy(&cuda_scene, GPU_cuda_scene, sizeof(struct scene), hipMemcpyDefault);
  cuda_scene.aabbs = aabbs;
  hipMemcpy(GPU_cuda_scene, &cuda_scene, sizeof(struct scene), hipMemcpyDefault);

# elif defined(PARTITIONING_OCTREE)

  // Octree creation must be done before the aabb creation,
  // as the aabb order are rewritten during the creation.

  struct octree *octree;
  struct AABB *aabb;

  create_octree(GPU_cuda_scene, &aabb, &octree);

  hipMemcpy(&cuda_scene, GPU_cuda_scene, sizeof(struct scene), hipMemcpyDefault);
  cuda_scene.octree = octree;
  cuda_scene.aabbs = aabb;
  hipMemcpy(GPU_cuda_scene, &cuda_scene, sizeof(struct scene), hipMemcpyDefault);

# endif

  /* End of Partitioning dependent code */

  return GPU_cuda_scene;
}
