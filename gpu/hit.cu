#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hit.h"
#include "vector3.h"

#include "partitioning/aabb.h"
#include "partitioning/octree.h"

__device__ static int ray_intersect(struct ray ray, vector3 *input_vertex, vector3 *input_normal,
                         vector3 *out, vector3 *normal)
{
  const float EPSILON = 0.0000001;
  vector3 vertex0 = input_vertex[0];
  vector3 vertex1 = input_vertex[1];
  vector3 vertex2 = input_vertex[2];
  vector3 normal0 = vector3_normalize(input_normal[0]);
  vector3 normal1 = vector3_normalize(input_normal[1]);
  vector3 normal2 = vector3_normalize(input_normal[2]);
  vector3 edge1, edge2, h, s, q;
  float a, f, u, v;
  edge1 = vector3_sub(vertex1, vertex0);
  edge2 = vector3_sub(vertex2, vertex0);
  h = vector3_cross(ray.direction, edge2);
  a = vector3_dot(edge1, h);
  if (a > -EPSILON && a < EPSILON)
    return 0;
  f = 1 / a;
  s = vector3_sub(ray.origin, vertex0);
  u = f * vector3_dot(s, h);
  if (u < 0.0 || u > 1.0)
    return 0;
  q = vector3_cross(s, edge1);
  v = f * vector3_dot(ray.direction, q);
  if (v < 0.0 || u + v > 1.0)
    return 0;

  float t = f * vector3_dot(edge2, q);
  if (t > EPSILON)
  {
    vector3 t2 = vector3_scale(vector3_normalize(ray.direction),
                                      t * vector3_length(ray.direction));
    *out = vector3_add(ray.origin, t2);
    *normal = vector3_add(vector3_add(vector3_scale(normal0, 1 - u - v),
                                    vector3_scale(normal1, u)),
                         vector3_scale(normal2, v));
    return 1;
  }
  return 0;
}

__device__ static struct ray triangle_collide(struct object object, struct ray ray)
{
  float distance = 0;
  struct ray ret = init_ray();
  for (size_t i = 0; i < object.triangle_count; i++)
  {
    vector3 out;
    vector3 normal;
    int has_intersected = ray_intersect(
      ray,
      get_vertex(object.triangles, i), get_normal(object.triangles, i),
      &out, &normal
    );

    if (has_intersected)
    {
      float new_dist = vector3_length(vector3_sub(out, ray.origin));
      if (new_dist > 0.01 && (new_dist < distance || distance == 0))
      {
        distance = new_dist;
        struct ray new_ret;
        new_ret.origin = out;
        new_ret.direction = normal;
        ret = new_ret;
      }
    }
  }
  return ret;
}


/* Partitioning dependent code */
# if defined(PARTITIONING_NONE) || defined(PARTITIONING_AABB)

__device__ struct ray collide(const struct scene* scene, struct ray ray, struct object* obj)
{
  float distance = 0;
  struct ray ret = init_ray();
  for (size_t i = 0; i < scene->object_count; i++)
  {
    #if defined(PARTITIONING_AABB)
    // Try the aabb first, to prevent checking for collision with all triangles
    // if there is no intersections.
    if (!hit_aabb(&scene->aabbs[i], &ray))
      continue;

    #endif


    struct ray new_ray = triangle_collide(scene->objects[i], ray);
    if (!vector3_is_zero(new_ray.direction))
    {
      float new_dist = vector3_length(vector3_sub(new_ray.origin, ray.origin));
      if (new_dist > 0.01 && (new_dist < distance || distance == 0))
      {
        distance = new_dist;
        ret = new_ray;
        *obj = scene->objects[i];
      }
    }


  }
  return ret;
}


# else /* PARTITIONING_OCTREE */


__device__ struct ray collide(const struct scene* scene, struct ray ray, struct object* obj)
{
  float distance = 0;
  struct ray ret = init_ray();

  constexpr int children_per_node = 8;
  constexpr int max_depth = 8;

  struct octree *octree_stack[children_per_node * max_depth];
  octree_stack[0] = scene->octree;
  size_t octree_stack_size = 1;


  while (octree_stack_size > 0)
  {
    struct octree current = *octree_stack[--octree_stack_size];
    if (hit_aabb(&current.box, &ray))
    {// It it's this octree, perform an intersection test on all it's objects, and add the children
      // Perform the intersection check on it's objects
      for (size_t i = current.start_index; i < current.end_index; ++i)
      {
        if (!hit_aabb(&scene->aabbs[i], &ray))
          continue;

        struct ray new_ray = triangle_collide(scene->objects[i], ray);
        if (!vector3_is_zero(new_ray.direction))
        {
          float new_dist = vector3_length(vector3_sub(new_ray.origin, ray.origin));
          if (new_dist > 0.01 && (new_dist < distance || distance == 0))
          {
            distance = new_dist;
            ret = new_ray;
            *obj = scene->objects[i];
          }
        }
      }

      // Add all of it's children
      for (size_t child = 0; child < children_per_node; ++child)
      {
        if (current.children[child] != nullptr)
        {
          octree_stack[octree_stack_size++] = current.children[child];
        }
      }
    }
  }

  return ret;
}

/*
__device__ struct ray collide(const struct scene* scene, struct ray ray, struct object* obj)
{
  float distance = 0;
  struct ray ret = init_ray();

  // Children per octree
  // Depth of the octree


  while (octree_stack_size > 0)
  {
    struct octree current = *octree_stack[--octree_stack_size];
    if (hit_aabb(&current.box, &ray))
    {// It it's this octree, perform an intersection test on all it's objects, and add the children

      // Perform the intersection check on it's objects
      for (size_t i = current.start_index; i < current.end_index; ++i)
      {
        struct ray new_ray = triangle_collide(scene->objects[i], ray);
        if (!vector3_is_zero(new_ray.direction))
        {
          float new_dist = vector3_length(vector3_sub(new_ray.origin, ray.origin));
          if (new_dist > 0.01 && (new_dist < distance || distance == 0))
          {
            distance = new_dist;
            ret = new_ray;
            *obj = scene->objects[i];
          }
        }
      }

      // Add all of it's children
      for (size_t child = 0; child < 8; ++child)
      {
        if (current.children[child] != nullptr)
        {
          octree_stack[++octree_stack_size] = current.children[child];
        }
      }
    }
  }

  return ret;
}
*/

# endif
/* End of Partitioning dependent code */



__device__ float3 operator+(const float3 &a, const float3 &b) {

  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);

}

__device__ float3 operator-(const float3 &a, const float3 &b) {

  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);

}


__device__ float operator~(const float3 &a) {
	return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}


__device__ float collide_dist(const struct scene* scene, struct ray ray)
{
  struct object object;
  struct ray new_ray = collide(scene, ray, &object);

  if (vector3_is_zero(new_ray.direction))
    return 0;
  else
  {
    vector3 res = vector3_sub(new_ray.origin, ray.origin);
    return vector3_length(res);
  }
}
