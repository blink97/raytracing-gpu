#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>
#include <functional>

#include "parser.h"
#include "partitioning/aabb.h"
#include "partitioning/octree.h"

#define TESTS_PATH "../../tests/"

// All tests
#define CUBE TESTS_PATH "cube.svati"
#define ISLAND_SMOOTH TESTS_PATH "island_smooth.svati" // High objects count
#define DARK_NIGHT TESTS_PATH "dark-night.svati" // Second highest objects count
#define SPHERES TESTS_PATH "spheres.svati"

#define FULL_BENCHMARK(function) \
  BENCHMARK_CAPTURE(function, simple_cube, CUBE); \
  BENCHMARK_CAPTURE(function, island_smooth, ISLAND_SMOOTH); \
  BENCHMARK_CAPTURE(function, dark_night, DARK_NIGHT); \
  BENCHMARK_CAPTURE(function, spheres, SPHERES);

/*
 * Benchmark the parser to see memory alignement difference
 */
void BM_parser(benchmark::State& st, const char *filename)
{
  for (auto _ : st)
    parser(filename);
}

FULL_BENCHMARK(BM_parser);


/*
 * Benchmark the AABB creation with the object strategy
 */
void BM_aabb_object(benchmark::State& st, const char *filename)
{
  struct scene scene = parser(filename);
  struct scene *cuda_scene = to_cuda(&scene);

  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * scene.object_count);

  // Compute the boundign box per objects.
  dim3 threadsPerBlock(32);
  dim3 numBlocks(ceil(scene.object_count * 1.0 / threadsPerBlock.x));

  for (auto _ : st)
    object_compute_bounding_box<<<numBlocks, threadsPerBlock>>>(cuda_scene, aabbs);

  hipFree(aabbs);
}

FULL_BENCHMARK(BM_aabb_object);


/*
 * Benchmark the scene scale finding with the basic strategy.
 */
void BM_find_scene_scale_basic(benchmark::State& st, const char *filename)
{
  struct scene scene = parser(filename);
  struct scene *cuda_scene = to_cuda(&scene);

  dim3 threadsPerBlock(32);
  dim3 numBlocks(ceil(scene.object_count * 1.0 / threadsPerBlock.x));

  // Compute the bounding box
  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * scene.object_count);
  object_compute_bounding_box<<<numBlocks, threadsPerBlock>>>(cuda_scene, aabbs);

  // Compute the global scale
  struct AABB *resulting_scale;
  hipMalloc(&resulting_scale, sizeof(struct AABB));

  for (auto _ : st)
    find_scene_scale_basic<<<numBlocks, threadsPerBlock>>>(aabbs, scene.object_count, resulting_scale);

  hipFree(aabbs);
}

FULL_BENCHMARK(BM_find_scene_scale_basic);


/*
 * Benchmark the scene scale finding with  the shared strategy.
 */
void BM_find_scene_scale_shared(benchmark::State& st, const char *filename)
{
  struct scene scene = parser(filename);
  struct scene *cuda_scene = to_cuda(&scene);

  dim3 threadsPerBlock(32);
  dim3 numBlocks(ceil(scene.object_count * 1.0 / threadsPerBlock.x));

  // Compute the bounding box
  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * scene.object_count);
  object_compute_bounding_box<<<numBlocks, threadsPerBlock>>>(cuda_scene, aabbs);

  // Compute the global scale
  struct AABB *resulting_scale;
  hipMalloc(&resulting_scale, sizeof(struct AABB));

  for (auto _ : st)
    find_scene_scale_shared<<<numBlocks, threadsPerBlock>>>(aabbs, scene.object_count, resulting_scale);

  hipFree(aabbs);
}

FULL_BENCHMARK(BM_find_scene_scale_shared);

/*
 * Benchmark the octree position creation.
 */
void BM_position_object(benchmark::State& st, const char *filename)
{
  struct scene scene = parser(filename);
  struct scene *cuda_scene = to_cuda(&scene);

  dim3 threadsPerBlock(32);
  dim3 numBlocks(ceil(scene.object_count * 1.0 / threadsPerBlock.x));

  // Compute the bounding box
  struct AABB *aabbs;
  hipMalloc(&aabbs, sizeof(struct AABB) * scene.object_count);
  object_compute_bounding_box<<<numBlocks, threadsPerBlock>>>(cuda_scene, aabbs);

  // Compute the global scale
  struct AABB *resulting_scale;
  hipMalloc(&resulting_scale, sizeof(struct AABB));

  find_scene_scale_shared<<<numBlocks, threadsPerBlock>>>(aabbs, scene.object_count, resulting_scale);

  octree_generation_position *positions;
  hipMalloc(&positions, sizeof(octree_generation_position) * scene.object_count);

  for (auto _ : st)
    position_object<<<numBlocks, threadsPerBlock>>>(aabbs, resulting_scale, positions, scene.object_count);

  hipFree(aabbs);
  hipFree(positions);
}

FULL_BENCHMARK(BM_position_object);


BENCHMARK_MAIN();