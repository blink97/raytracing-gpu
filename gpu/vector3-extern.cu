#include "hip/hip_runtime.h"
#include <math.h>

#include "vector3.h"

__host__ __device__ float vector3_dot(vector3 a, vector3 b)
{
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ float vector3_length(vector3 a)
{
  return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__host__ __device__ int vector3_cmp(vector3 a, vector3 b)
{
  return a.x == b.x && a.y == b.y && a.z == b.z;
}

__host__ __device__ int vector3_is_zero(vector3 a)
{
  return a.x == 0 && a.y == 0 && a.z == 0;
}
